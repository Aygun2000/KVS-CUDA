#include <stdio.h>
#include <hip/hip_runtime.h>
#include <chrono>
#include "kv_store.h"
#include <cstdio>

void run_tests() {
    const int n = 100000;

    uint32_t* keys = new uint32_t[n];
    uint32_t* values = new uint32_t[n];
    uint32_t* query_keys = new uint32_t[n];
    uint32_t* results = new uint32_t[n];

    for (int i = 0; i < n; ++i) {
        keys[i] = i;
        values[i] = i * 10;
        query_keys[i] = i; 
    }

    uint32_t* d_keys, * d_values, * d_query_keys, * d_results;
    uint32_t* hash_table_keys, * hash_table_values;

    hipMalloc(&d_keys, n * sizeof(uint32_t));
    hipMalloc(&d_values, n * sizeof(uint32_t));
    hipMalloc(&d_query_keys, n * sizeof(uint32_t));
    hipMalloc(&d_results, n * sizeof(uint32_t));
    hipMalloc(&hash_table_keys, TABLE_SIZE * sizeof(uint32_t));
    hipMalloc(&hash_table_values, TABLE_SIZE * sizeof(uint32_t));

    hipMemcpy(d_keys, keys, n * sizeof(uint32_t), hipMemcpyHostToDevice);
    hipMemcpy(d_values, values, n * sizeof(uint32_t), hipMemcpyHostToDevice);
    hipMemcpy(d_query_keys, query_keys, n * sizeof(uint32_t), hipMemcpyHostToDevice);
    hipMemset(hash_table_keys, UINT32_MAX, TABLE_SIZE * sizeof(uint32_t));
    hipMemset(hash_table_values, 0, TABLE_SIZE * sizeof(uint32_t));

    // GPU Timing Events for Insert
    hipEvent_t insertStart, insertStop, lookupStart, lookupStop;
    hipEventCreate(&insertStart);
    hipEventCreate(&insertStop);
    hipEventCreate(&lookupStart);
    hipEventCreate(&lookupStop);
    float elapsed_time;

    // **GPU Insert Timing**
    hipEventRecord(insertStart);
    kv_insert << <(n + 255) / 256, 256 >> > (d_keys, d_values, hash_table_keys, hash_table_values, n);
    hipError_t err = hipGetLastError();  
    if (err != hipSuccess) {
        printf("CUDA Error (Insert Launch): %s\n", hipGetErrorString(err));
    }
    hipEventRecord(insertStop);
    hipEventSynchronize(insertStop);
    hipEventElapsedTime(&elapsed_time, insertStart, insertStop);
    printf("GPU Insert Time: %f ms\n", elapsed_time);

    // **GPU Lookup Timing**
    hipEventRecord(lookupStart);
    kv_lookup << <(n + 255) / 256, 256 >> > (d_query_keys, d_results, hash_table_keys, hash_table_values, n);
    err = hipGetLastError();
    if (err != hipSuccess) {
        printf("CUDA Error (Lookup Launch): %s\n", hipGetErrorString(err));
    }
    hipEventRecord(lookupStop);
    hipEventSynchronize(lookupStop);
    hipEventElapsedTime(&elapsed_time, lookupStart, lookupStop);
    printf("GPU Lookup Time: %f ms\n", elapsed_time);

    // Free CUDA events
    hipEventDestroy(insertStart);
    hipEventDestroy(insertStop);
    hipEventDestroy(lookupStart);
    hipEventDestroy(lookupStop);

    // Free memory
    hipFree(d_keys);
    hipFree(d_values);
    hipFree(d_query_keys);
    hipFree(d_results);
    hipFree(hash_table_keys);
    hipFree(hash_table_values);

    delete[] keys;
    delete[] values;
    delete[] query_keys;
    delete[] results;
}

#ifdef GPU_TEST
int main() {
    printf("Starting GPU Key-Value Store Test...\n");
    run_tests();
    printf("GPU Test Completed.\n");
    return 0;
}
#endif
